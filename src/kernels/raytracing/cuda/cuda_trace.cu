#include "hip/hip_runtime.h"
/*
 * Tracy, a simple raytracer
 * inspired by "Ray Tracing in One Weekend" minibooks
 *
 * (c) Carlo Casta, 2017-2021
 */
#include "cuda_trace.cuh"
#include "cuda_details.h"
#include "cuda_log.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>
#include <hiprand.h>

#include "collision.h"
#include "camera.h"
#include "scene.h"

constexpr uint32_t kMaxBounces{ TRACY_MAX_BOUNCES };

__device__ bool Intersects(const Ray& ray, const KernelData& data, HitData& intersection)
{
    bool hit_any_mesh{ false };

    const vec3 ray_origin{ ray.GetOrigin() };
    const vec3 ray_invdir{ ray.GetDirectionInverse() };

    for (uint32_t i = 0; i < data.GetMeshCount(); ++i)
    {
        auto& object = data.GetMesh(i);
        auto& aabb = object.GetAABB();

        if (collision::RayAABB(ray_origin, ray_invdir, aabb.minbound, aabb.maxbound, intersection.t))
        {
            collision::MeshHitData mesh_hit(intersection.t);
            if (collision::RayMesh(ray, object, mesh_hit))
            {
                intersection.t = mesh_hit.RayT;
                intersection.uv = mesh_hit.TriangleUV;
                intersection.triangle_index = mesh_hit.TriangleIndex;
                intersection.object_index = i;
                hit_any_mesh = true;
            }
        }
    }

    if (hit_any_mesh)
    {
        const Mesh& mesh = data.GetMesh(intersection.object_index);
    
        const Index i0 = mesh.GetIndex(intersection.triangle_index + 0);
        const Index i1 = mesh.GetIndex(intersection.triangle_index + 1);
        const Index i2 = mesh.GetIndex(intersection.triangle_index + 2);
    
        const Vertex v0 = mesh.GetVertex(i0);
        const Vertex v1 = mesh.GetVertex(i1);
        const Vertex v2 = mesh.GetVertex(i2);
    
        const vec2 uv = intersection.uv;
    
        intersection.point = ray.GetPoint(intersection.t);
        intersection.normal = normalize((1.f - uv.x - uv.y) * v0.normal + uv.x * v1.normal + uv.y * v2.normal);
        intersection.tangent = (1.f - uv.x - uv.y) * v0.tangent + uv.x * v1.tangent + uv.y * v2.tangent;
        intersection.uv = (1.f - uv.x - uv.y) * v0.uv0 + uv.x * v1.uv0 + uv.y * v2.uv0;
        intersection.material = mesh.GetMaterial();
    }
    
    return hit_any_mesh;
}


__device__ vec3 Trace(Ray&& ray, const KernelData& data, RandomCtx random_ctx)
{
    Ray current_ray{ std::move(ray) };
    vec3 throughput{ 1.f, 1.f, 1.f };
    vec3 pixel;

    uint32_t raycount{};

    for (uint32_t t = 0; t < kMaxBounces; ++t)
    {
        ++raycount;

        HitData intersection_data;
        intersection_data.t = FLT_MAX;

        vec3 attenuation;
        vec3 emission;

        if (Intersects(current_ray, data, intersection_data))
        {

#if DEBUG_SHOW_BASECOLOR
            return data.GetMaterial(intersection_data.material).GetBaseColor(data, intersection_data);
#elif DEBUG_SHOW_NORMALS
            return .5f * normalize((1.f + mat3(data.GetCamera().GetView()) * data.GetMaterial(intersection_data.material).GetNormal(data, intersection_data)));
#elif DEBUG_SHOW_METALNESS
            return vec3(data.GetMaterial(intersection_data.material).GetMetalness(data, intersection_data));
#elif DEBUG_SHOW_ROUGHNESS
            return vec3(data.GetMaterial(intersection_data.material).GetRoughness(data, intersection_data));
#elif DEBUG_SHOW_EMISSIVE
            return data.GetMaterial(intersection_data.material).GetEmissive(data, intersection_data);
#endif

            data.GetMaterial(intersection_data.material).Scatter(data, current_ray, intersection_data, attenuation, emission, current_ray, random_ctx);
            {
                pixel += emission * throughput;
                throughput *= attenuation;
            }
        }
        else
        {
            const vec3 v{ current_ray.GetDirection() };
            intersection_data.uv = vec2(atan2f(v.z, v.x) / (2 * PI), asinf(v.y) / PI) + 0.5f;
            emission = data.GetMaterial(Scene::SKY_MATERIAL_ID).GetEmissive(data, intersection_data);

            pixel += emission * throughput;
            break;
        }

#if USE_RUSSIAN_ROULETTE
        float p = EPS + max(throughput.r, max(throughput.g, throughput.b));
        if (fastrand(random_ctx) > p)
        {
            break;
        }

        throughput *= rcp(p);
#endif
    }

    atomicAdd(data.raycount_, raycount);
    return pixel;
}

//
// Kernels
//
__global__ void TraceKernel(hipSurfaceObject_t surface, KernelData data, uint32_t w, uint32_t h, uint32_t frame_count)
{
    const uint32_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
    const uint32_t j = (blockIdx.y * blockDim.y) + threadIdx.y;
    const uint32_t idx = j * w + i;

    if LIKELY(i < w && j < h)
    {
        RandomCtxData hiprand = data.randctx_[idx];
        RandomCtx curand_ctx = &hiprand;

        const float s = ((i + fastrand(curand_ctx)) / static_cast<float>(w));
        const float t = ((j + fastrand(curand_ctx)) / static_cast<float>(h));
        const vec4 color{ Trace(data.GetCamera().GetRayFrom(s, t), data, curand_ctx), 1.f };
        
        data.randctx_[idx] = hiprand;

#if ACCUMULATE_SAMPLES

        const float blend_factor{ frame_count / (frame_count + 1.f) };
        
        const vec4 prev_color{ surf2Dread<float4>(surface, i * sizeof(float4), j) };

        surf2Dwrite<float4>(lerp(color, prev_color, blend_factor), surface, i * sizeof(float4), j);

#else

        surf2Dwrite<float4>(color, surface, i * sizeof(float4), j);

#endif
    }
}

__global__ void InitRandom(KernelData data, uint32_t w, uint32_t h)
{
    const uint32_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
    const uint32_t j = (blockIdx.y * blockDim.y) + threadIdx.y;
    const uint32_t idx = j * w + i;

    if LIKELY(i < w && j < h)
    {
        hiprand_init(0x12345, idx, 0, &data.randctx_[idx]);
    }
}
//
// End Kernels
//

void CUDATraceKernel::Trace()
{
    CUDAAssert(hipGraphicsMapResources(1, &host_data_.output_resource, 0));
    
    hipArray_t output_array{};
    CUDAAssert(hipGraphicsSubResourceGetMappedArray(&output_array, host_data_.output_resource, 0, 0));
    
    hipResourceDesc description{};
    description.resType = hipResourceTypeArray;
    description.res.array.array = output_array;
    
    hipSurfaceObject_t surface_object;
    CUDAAssert(hipCreateSurfaceObject(&surface_object, &description));
    
    TraceKernel<<<host_data_.grid, host_data_.block>>>(surface_object, kernel_data_, host_data_.width, host_data_.height, host_data_.frame_counter_++);

    CUDAAssert(hipDestroySurfaceObject(surface_object));
    
    CUDAAssert(hipGraphicsUnmapResources(1, &host_data_.output_resource, 0));
    
    // TODO: find something better to keep track of raycount
    uint32_t raycount;
    CUDAAssert(hipMemcpy(&raycount, kernel_data_.raycount_, sizeof(uint32_t), hipMemcpyDeviceToHost));
    host_data_.raycount += raycount;

    CUDAAssert(hipStreamSynchronize(0));

    CUDAAssert(hipMemsetAsync(kernel_data_.raycount_, 0, sizeof(uint32_t)));
}

void CUDATraceKernel::UpdateCamera(const Camera& in_Camera)
{
    host_data_.frame_counter_ = 0;
    CUDAAssert(hipMemcpy(kernel_data_.camera_, &in_Camera, sizeof(Camera), hipMemcpyHostToDevice));
}

bool CUDATraceKernel::Setup(RenderData* in_RenderData)
{
    if (in_RenderData)
    {
        const uint32_t w{ in_RenderData->width };
        const uint32_t h{ in_RenderData->height };
        const dim3 block{ 8, 8 };
        const dim3 grid{ (w + block.x - 1) / block.x, (h + block.y - 1) / block.y };

        host_data_.width = w;
        host_data_.height = h;
        host_data_.block = block;
        host_data_.grid = grid;

        CUDAAssert(hipGraphicsGLRegisterImage(&host_data_.output_resource,
                                               in_RenderData->output_texture,
                                               GL_TEXTURE_2D,
                                               hipGraphicsRegisterFlagsSurfaceLoadStore));

        // rand seed generation on device
        CUDAAssert(hipMalloc(&kernel_data_.randctx_, host_data_.width * host_data_.height * sizeof(hiprandState)));

        InitRandom<<<host_data_.grid, host_data_.block>>>(kernel_data_, host_data_.width, host_data_.height);

        CUDAAssert(hipMalloc(&kernel_data_.raycount_, sizeof(uint32_t)));
        CUDAAssert(hipMemset(kernel_data_.raycount_, 0, sizeof(uint32_t)));
        
        CUDAAssert(hipDeviceSynchronize());

        return true;
    }

    return false;
}

bool CUDATraceKernel::SetupScene(const Scene& in_Scene)
{
    CUDAAssert(hipMalloc(&kernel_data_.camera_, sizeof(Camera)));
    CUDAAssert(hipMemcpy(kernel_data_.camera_, &in_Scene.GetCamera(), sizeof(Camera), hipMemcpyHostToDevice));

    kernel_data_.meshcount_ = in_Scene.GetObjectCount();
    CUDAAssert(hipMalloc(&kernel_data_.meshes_, kernel_data_.meshcount_ * sizeof(Mesh)));
    for (uint32_t i = 0; i < kernel_data_.meshcount_; ++i)
    {
        auto& host_mesh = in_Scene.GetObject(i);

        Vertex* vertices{};
        uint32_t vertexcount{ host_mesh.GetVertexCount() };
        CUDAAssert(hipMalloc(&vertices, vertexcount * sizeof(Vertex)));
        CUDAAssert(hipMemcpy(vertices, host_mesh.GetVertices(), vertexcount * sizeof(Vertex), hipMemcpyHostToDevice));

        Index* indices{};
        uint32_t indexcount{ host_mesh.GetIndexCount() };
        CUDAAssert(hipMalloc(&indices, indexcount * sizeof(Index)));
        CUDAAssert(hipMemcpy(indices, host_mesh.GetIndices(), indexcount * sizeof(Index), hipMemcpyHostToDevice));

        Mesh* mesh = new Mesh(vertices, vertexcount, indices, indexcount, host_mesh.GetAABB(), host_mesh.GetMaterial());
        CUDAAssert(hipMemcpy(&kernel_data_.meshes_[i], mesh, sizeof(Mesh), hipMemcpyHostToDevice));
    }

    kernel_data_.materialcount_ = static_cast<uint32_t>(in_Scene.GetMaterials().size());
    CUDAAssert(hipMalloc(&kernel_data_.materials_, kernel_data_.materialcount_ * sizeof(Material)));
    CUDAAssert(hipMemcpy(kernel_data_.materials_, in_Scene.GetMaterials().data(), kernel_data_.materialcount_ * sizeof(Material), hipMemcpyHostToDevice));

    kernel_data_.texturecount_ = static_cast<uint32_t>(in_Scene.GetTextures().size());
    CUDAAssert(hipMalloc(&kernel_data_.textures_, kernel_data_.texturecount_ * sizeof(Texture)));
    for (uint32_t i = 0; i < kernel_data_.texturecount_; ++i)
    {
        auto&& host_texture = in_Scene.GetTexture(i);
        uint32_t host_texture_size = host_texture.GetWidth() * host_texture.GetHeight();

        vec4* pixels{};
        CUDAAssert(hipMalloc(&pixels, host_texture_size * sizeof(vec4)));
        CUDAAssert(hipMemcpy(pixels, host_texture.GetPixels(), host_texture_size * sizeof(vec4), hipMemcpyHostToDevice));

        Texture* texture = new Texture(host_texture.GetWidth(), host_texture.GetHeight(), pixels);
        CUDAAssert(hipMemcpy(&kernel_data_.textures_[i], texture, sizeof(Texture), hipMemcpyHostToDevice));
    }

    CUDAAssert(hipDeviceSynchronize());

    return true;
}

void CUDATraceKernel::Shutdown()
{
    CUDAAssert(hipGraphicsUnregisterResource(host_data_.output_resource));

    CUDAAssert(hipFree(kernel_data_.raycount_));

    CUDAAssert(hipFree(kernel_data_.randctx_));

    CUDAAssert(hipFree(kernel_data_.textures_));

    CUDAAssert(hipFree(kernel_data_.materials_));

    CUDAAssert(hipFree(kernel_data_.meshes_));

    CUDAAssert(hipFree(kernel_data_.camera_));
}
