#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#if defined(__HIPCC__)
 #define CUDA_CALL __host__ __device__
#else
 #define CUDA_CALL
#endif

#include "common.h"
#include "log.h"

#include "ray.h"
#include "camera.h"
#include "material.h"
#include "cuda_mesh.h"
#include "scene.h"
#include "cuda_scene.h"

constexpr int MAX_GPU = 32;
constexpr int MAX_DEPTH = 5;

struct Color
{
	static_assert(sizeof(uint32_t) == 4 * sizeof(uint8_t), "u32 != 4 * u8 :/");

	__device__ constexpr Color()                 : rgba(0)       {}
	__device__ constexpr Color(uint32_t in_rgba) : rgba(in_rgba) {}

	union
	{
		struct
		{
			uint8_t r;
			uint8_t g;
			uint8_t b;
			uint8_t a;
		};
		
		uint32_t rgba;
	};
};

__device__ constexpr inline uint32_t ToInt(vec3 color)
{
	color *= 255.99f;

	Color c;
    c.r = static_cast<uint8_t>(clamp(color.r, 0.0f, 255.0f));
    c.g = static_cast<uint8_t>(clamp(color.g, 0.0f, 255.0f));
    c.b = static_cast<uint8_t>(clamp(color.b, 0.0f, 255.0f));
	c.a = 255;

    return c.rgba;
}

__device__ constexpr inline vec3 ToFloat(uint32_t color)
{
	Color c;
	c.rgba = color;
	
	return vec3{ c.r / 255.f, c.g / 255.f, c.b / 255.f };
}

__device__ inline float fastrand(hiprandState* curand_ctx)
{
    return hiprand_uniform(curand_ctx);
}

__device__ bool IntersectsWithBoundingBox(const BBox& box, const Ray& ray, float nearest_intersection = FLT_MAX)
{
    const vec3 inv_ray = ray.GetInvDirection();
    const vec3 minbound = (box.minbound - ray.GetOrigin()) * inv_ray;
    const vec3 maxbound = (box.maxbound - ray.GetOrigin()) * inv_ray;

    vec3 tmin1 = pmin(minbound, maxbound);
    vec3 tmax1 = pmax(minbound, maxbound);

    float tmin = max(tmin1.x, max(tmin1.y, tmin1.z));
    float tmax = min(tmax1.x, min(tmax1.y, tmax1.z));

    return (tmax >= max(1.e-8f, tmin) && tmin < nearest_intersection);
}

__device__ bool IntersectsWithMesh(const CUDAMesh& mesh, const Ray& in_ray, HitData& inout_intersection)
{
    bool hit_triangle = false;

    if (IntersectsWithBoundingBox(mesh.aabb_, in_ray, inout_intersection.t))
    {
        vec3 ray_direction = in_ray.GetDirection();
        vec3 ray_origin = in_ray.GetOrigin();

        int tris = mesh.indexcount_ / 3;
        for (int i = 0; i < tris; ++i)
        {
            const Index i0 = mesh.indices_[i * 3 + 0];
            const Index i1 = mesh.indices_[i * 3 + 1];
            const Index i2 = mesh.indices_[i * 3 + 2];

            const vec3 v0 = mesh.vertices_[i0].pos;
            const vec3 v1 = mesh.vertices_[i1].pos;
            const vec3 v2 = mesh.vertices_[i2].pos;

            const vec3 v0v1 = v1 - v0;
            const vec3 v0v2 = v2 - v0;

            vec3 pvec = cross(ray_direction, v0v2);
            float det = dot(v0v1, pvec);

            // if the determinant is negative the triangle is backfacing
            // if the determinant is close to 0, the ray misses the triangle
            if (det < 1.e-8f)
            {
                continue;
            }

            float invDet = 1.f / det;

            vec3 tvec = ray_origin - v0;
            float u = dot(tvec, pvec) * invDet;
            if (u < .0f || u > 1.f)
            {
                continue;
            }

            vec3 qvec = cross(tvec, v0v1);
            float v = dot(ray_direction, qvec) * invDet;
            if (v < .0f || u + v > 1.f)
            {
                continue;
            }

            float t = dot(v0v2, qvec) * invDet;
            if (t < inout_intersection.t && t > 1.e-3f)
            {
                inout_intersection.t = dot(v0v2, qvec) * invDet;
                inout_intersection.uv = vec2{ u, v };
                inout_intersection.triangle_index = i * 3;
                hit_triangle = true;
            }
        }
    }

    return hit_triangle;
}

__device__ bool ComputeIntersection(CUDAMesh* in_objects, int objectcount, const Ray& ray, HitData& intersection_data)
{
    bool hit_any_mesh = false;
    
    for (int i = 0; i < objectcount; ++i)
    {
        if (IntersectsWithMesh(in_objects[i], ray, intersection_data))
        {
            intersection_data.object_index = i;
            hit_any_mesh = true;
        }
    }

    if (hit_any_mesh)
    {
        const CUDAMesh& m = in_objects[intersection_data.object_index];
        const Vertex v0 = m.vertices_[m.indices_[intersection_data.triangle_index + 0]];
        const Vertex v1 = m.vertices_[m.indices_[intersection_data.triangle_index + 1]];
        const Vertex v2 = m.vertices_[m.indices_[intersection_data.triangle_index + 2]];

        intersection_data.point = ray.GetPoint(intersection_data.t);
        intersection_data.normal = (1.f - intersection_data.uv.x - intersection_data.uv.y) * v0.normal + intersection_data.uv.x * v1.normal + intersection_data.uv.y * v2.normal;
        intersection_data.uv = (1.f - intersection_data.uv.x - intersection_data.uv.y) * v0.uv0 + intersection_data.uv.x * v1.uv0 + intersection_data.uv.y * v2.uv0;
        intersection_data.material = &m.material_;
    }

    return hit_any_mesh;
}

__device__ inline vec3 TraceInternal(const Camera& in_camera, const Ray& in_ray, CUDAMesh* in_objects, int objectcount, int& inout_raycount)
{
    vec3 current_color = { 1.f, 1.f, 1.f };
    Ray current_ray = { in_ray };

    HitData hit_data;
    hit_data.t = FLT_MAX;

    for (int i = 0; i < MAX_DEPTH; ++i)
    {
        ++inout_raycount;

        if (ComputeIntersection(in_objects, objectcount, current_ray, hit_data))
        {

#if DEBUG_SHOW_NORMALS
            return .5f * normalize(1.f + (mat3(in_camera.GetView()) * hit_data.normal));
#else
            Ray scattered;
            vec3 attenuation;
            vec3 emission;
            if (hit_data.material->Scatter(current_ray, hit_data, attenuation, emission, scattered))
            {
                current_color *= attenuation;
                current_ray = scattered;
            }
            else
            {
                current_color *= emission;
                return current_color;
            }
#endif
        }
        else
        {
            return {};
        }
    }

    return {};
}

__global__ void Trace(Camera* in_camera,
                      CUDAMesh* in_objects,
                      int in_objectcount,
                      uint32_t* output,
                      int width,
                      int height,
                      hiprandState* rand_state,
                      int* raycount,
                      int framecount)
{
    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int j = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (i >= width || j >= height)
    {
        return;
    }

    const float f_width = static_cast<float>(width);
    const float f_height = static_cast<float>(height);

    hiprandState curand_ctx = rand_state[j * width + i];
    int old_raycount = raycount[j * width + i];

    int cur_raycount = 0;

    vec3 cur_color{};
    for (int sample = 0; sample < 1; ++sample)
    {
        float s = ((i + fastrand(&curand_ctx)) / f_width);
        float t = ((j + fastrand(&curand_ctx)) / f_height);
    
        Ray r = in_camera->GetRayFrom(s, t);
        cur_color += TraceInternal(*in_camera, r, in_objects, in_objectcount, cur_raycount);
    }
    
    rand_state[j * width + i] = curand_ctx;
    raycount[j * width + i] = old_raycount + cur_raycount;

	const float blend_factor = framecount / static_cast<float>(framecount + 1);
    vec3 old_color = ToFloat(output[j * width + i]);
    output[j * width + i] = ToInt(lerp(cur_color, old_color, blend_factor));
}

__global__ void InitRandom(hiprandState* rand_state, int width, int height)
{
    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int j = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (i >= width || j >= height)
    {
        return;
    }

    hiprand_init(clock64(), i, j, &rand_state[j * width + i]);
}

extern "C" void cuda_setup(const Scene& in_scene, CUDAScene* out_scene)
{
    int num_gpus = 0;
    CUDAAssert(hipGetDeviceCount(&num_gpus));
    
    hipDeviceProp_t gpu_properties[MAX_GPU];
    for (int i = 0; i < num_gpus; i++)
    {
        CUDAAssert(hipGetDeviceProperties(&gpu_properties[i], i));

        CUDALog("Device %d (%s):\n"
                "\t%d threads\n"
                "\tblocksize: %dx%dx%d\n"
                "\tshmem per block: %dKb\n"
                "\tgridsize: %dx%dx%d\n\n",
                i,
                gpu_properties[i].name,
                gpu_properties[i].maxThreadsPerBlock,
                gpu_properties[i].maxThreadsDim[0], gpu_properties[i].maxThreadsDim[1], gpu_properties[i].maxThreadsDim[2],
                gpu_properties[i].sharedMemPerBlock / 1024,
                gpu_properties[i].maxGridSize[0], gpu_properties[i].maxGridSize[1], gpu_properties[i].maxGridSize[2]);
    }

    CUDAAssert(hipSetDevice(0));

    //
    // copy data to the device
    //

    CUDAAssert(hipMalloc(&out_scene->objects_, in_scene.GetObjectCount() * sizeof(CUDAMesh)));
    for (int i = 0; i < in_scene.GetObjectCount(); ++i)
    {
        CUDAMesh cmesh(in_scene.GetObject(i));
        CUDAAssert(hipMemcpy(&out_scene->objects_[i], &cmesh, sizeof(CUDAMesh), hipMemcpyHostToDevice));
    }
    out_scene->objectcount_ = in_scene.GetObjectCount();

    CUDAAssert(hipMalloc(&out_scene->d_camera_, sizeof(Camera)));
    CUDAAssert(hipMemcpy(out_scene->d_camera_, &in_scene.GetCamera(), sizeof(Camera), hipMemcpyHostToDevice));

    CUDAAssert(hipMalloc(&out_scene->d_rand_state, out_scene->width * out_scene->height * sizeof(hiprandState)));

    dim3 block(16, 16, 1);
    dim3 grid(out_scene->width / block.x + 1, out_scene->height / block.y + 1, 1);
    InitRandom<<<grid, block>>> (out_scene->d_rand_state, out_scene->width, out_scene->height);

    out_scene->h_raycount = new int[out_scene->width * out_scene->height];
    CUDAAssert(hipMalloc(&out_scene->d_raycount, out_scene->width * out_scene->height * sizeof(int)));
    CUDAAssert(hipMemset(out_scene->d_raycount, 0, out_scene->width * out_scene->height * sizeof(int)));
}


extern "C" void cuda_trace(CUDAScene* scene, unsigned int* output, int framecount)
{
    CUDAAssert(hipSetDevice(0));

    dim3 block(16, 16, 1);
    dim3 grid(scene->width / block.x + 1, scene->height / block.y + 1, 1);

    Trace<<<grid, block>>>(scene->d_camera_, scene->objects_, scene->objectcount_, output, scene->width, scene->height, scene->d_rand_state, scene->d_raycount, framecount);
    
    CUDAAssert(hipGetLastError());
}
