#include "hip/hip_runtime.h"
/*
 * Tracy, a simple raytracer
 * inspired by "Ray Tracing in One Weekend" minibooks
 *
 * (c) Carlo Casta, 2018
 */
#include <cstdio>
#include <cfloat>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "mathutils.cuh"


#ifndef NODEBUG
#define CUDALOG(...) printf(__VA_ARGS__)
#else 
#define CUDALOG(...) do {} while(0);
#endif

//
// from hip/hip_runtime_api.h
// NVidia CUDA samples
// 
template <typename T>
void check(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        hipError_t cuda_error = hipGetLastError();

        CUDALOG("[CUDA error] at %s:%d code=%d (%s) \"%s\" \n", file, line, static_cast<unsigned int>(result), hipGetErrorName(cuda_error), func);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

//
// ----------------------------------------------------------------------------
//

__device__ float schlick_fresnel(float costheta, float ior)
{
    float r0 = (1.f - ior) / (1.f + ior);
    r0 *= r0;
    return r0 + (1.f - r0) * powf(max(.0f, (1.f - costheta)), 5);
}

__device__ hiprandState curand_ctx;
__device__ float cuda_fastrand()
{
    return hiprand_uniform(&curand_ctx);
}

__device__ float3 cuda_random_on_unit_sphere()
{
    float z = cuda_fastrand() * 2.f - 1.f;
    float a = cuda_fastrand() * 2.f * PI;
    float r = sqrtf(max(.0f, 1.f - z * z));

    return make_float3(r * cosf(a), r * sinf(a), z);
}

//
// ----------------------------------------------------------------------------
//

struct DRay
{
    float3 origin;
    float3 direction;

    __device__ float3 point_at(float t) const { return origin + t * direction; }
};

struct DMaterial;
struct DIntersection
{
    int type;
    int index;
    float t;
    float3 point;
    float3 normal;
    DMaterial* material;
};

enum { eLAMBERTIAN, eMETAL, eDIELECTRIC, eEMISSIVE };
struct DMaterial
{
    int type;
    float3 albedo;
    float roughness;
    float ior;

    __device__ bool scatter(const DRay& ray, const DIntersection& hit, float3& attenuation, float3& emission, DRay& scattered)
    {
        if (type == eLAMBERTIAN)
        {
            float3 target = hit.point + hit.normal + cuda_random_on_unit_sphere();
            scattered.origin = hit.point;
            scattered.direction = normalize(target - hit.point);
            attenuation = albedo;
            emission = make_float3(.0f, .0f, .0f);

            return true;
        }
        else if (type == eMETAL)
        {
            float3 reflected = reflect(normalize(ray.direction), hit.normal);
            scattered.origin = hit.point;
            scattered.direction = reflected + roughness * cuda_random_on_unit_sphere();

            attenuation = albedo;
            emission = make_float3(.0f, .0f, .0f);
            
            return (dot(scattered.direction, hit.normal) > .0f);
        }
        else if (type == eDIELECTRIC)
        {
            float3 outward_normal;
            attenuation = { 1.f, 1.f, 1.f };
            emission = make_float3(.0f, .0f, .0f);

            float ni_nt;
            float cosine;
            if (dot(ray.direction, hit.normal) > .0f)
            {
                outward_normal = -1.f * hit.normal;
                ni_nt = ior;
                cosine = dot(ray.direction, hit.normal) / length(ray.direction);
                cosine = sqrtf(1.f - ior * ior * (1.f - cosine - cosine));
            }
            else
            {
                outward_normal = hit.normal;
                ni_nt = 1.f / ior;
                cosine = -dot(ray.direction, hit.normal) / length(ray.direction);
            }

            float3 refracted;
            bool is_refracted = refract(ray.direction, outward_normal, ni_nt, refracted);
            float reflect_chance = (is_refracted) ? schlick_fresnel(cosine, ior) : 1.0f;
            
            scattered.origin = hit.point;
            scattered.direction = normalize((cuda_fastrand() < reflect_chance) ? reflect(ray.direction, hit.normal) : refracted);

            return true;
        }
        else if (type == eEMISSIVE)
        {
            emission = albedo;
            return false;
        }

        return false;
    }
};

struct DSphere
{
    float3 center;
    float radius;
    DMaterial material;

    __device__ void hit_data(const DRay& ray, DIntersection& hit)
    {
        hit.point = ray.point_at(hit.t);
        hit.normal = (hit.point - center) / radius;
        hit.material = &material;
    }
};

struct DBox
{
    float3 min_limit;
    float3 max_limit;
    float3 rot;
    DMaterial material;

    __device__ void hit_data(const DRay& ray, DIntersection& hit)
    {
        hit.point = ray.point_at(hit.t);
        hit.normal = normal(hit.point);
        hit.material = &material;
    }

    __device__ float3 normal(const float3& point)
    {
        if (fabs(min_limit.x - point.x) < EPS) return make_float3(-1.f,  .0f,  .0f);
        if (fabs(max_limit.x - point.x) < EPS) return make_float3( 1.f,  .0f,  .0f);
        if (fabs(min_limit.y - point.y) < EPS) return make_float3( .0f, -1.f,  .0f);
        if (fabs(max_limit.y - point.y) < EPS) return make_float3( .0f,  1.f,  .0f);
        if (fabs(min_limit.z - point.z) < EPS) return make_float3( .0f,  .0f, -1.f);
        return make_float3(.0f, .0f, 1.f);
    }
};

enum { eSPHERE, eBOX };
__device__ bool intersect_spheres(DRay ray, DSphere* spheres, int sphere_count, DIntersection& hit_data)
{
    bool hit_something = false;

    for (int i = 0; i < sphere_count; ++i)
    {
        DSphere& sphere = spheres[i];

        float3 oc = ray.origin - sphere.center;
        float b = dot(oc, ray.direction);
        float c = dot(oc, oc) - sphere.radius * sphere.radius;

        if (b <= .0f || c <= .0f)
        {
            float discriminant = b * b - c;
            if (discriminant > 0.f)
            {
                discriminant = sqrtf(discriminant);

                float t0 = -b - discriminant;
                if (t0 > EPS && t0 < hit_data.t)
                {
                    hit_data.t = t0;
                    hit_data.type = eSPHERE;
                    hit_data.index = i;
                    hit_something = true;
                }

                float t1 = -b + discriminant;
                if (t1 > EPS && t1 < hit_data.t)
                {
                    hit_data.t = t1;
                    hit_data.type = eSPHERE;
                    hit_data.index = i;
                    hit_something = true;
                }
            }
        }
    }

    return hit_something;
}

__device__ bool intersect_boxes(DRay ray, DBox* boxes, int box_count, DIntersection& hit_data)
{
    bool hit_something = false;

    for (int i = 0; i < box_count; ++i)
    {
        DBox& box = boxes[i];

        float tmin = EPS;
        float tmax = FLT_MAX;

        bool boxhit = false;
        for (int side = 0; side < 3; ++side)
        {
            // TODO: think something better
            float direction = (side == 0) ? ray.direction.x : (side == 1) ? ray.direction.y : ray.direction.z;
            float origin = (side == 0) ? ray.origin.x : (side == 1) ? ray.origin.y : ray.origin.z;
            float minbound = (side == 0) ? box.min_limit.x : (side == 1) ? box.min_limit.y : box.min_limit.z;
            float maxbound = (side == 0) ? box.max_limit.x : (side == 1) ? box.max_limit.y : box.max_limit.z;

            if (fabs(direction) < EPS)
            {
                if (origin < minbound || origin > maxbound) { boxhit = false; break; }
            }
            else
            {
                float ood = 1.f / direction;
                float t1 = (minbound - origin) * ood;
                float t2 = (maxbound - origin) * ood;

                if (t1 > t2) swap(t1, t2);

                tmin = max(tmin, t1);
                tmax = min(tmax, t2);

                if (tmin > tmax || tmin > hit_data.t) { boxhit = false; break; }
                boxhit = true;
            }
        }

        if (boxhit)
        {
            hit_data.t = tmin;
            hit_data.type = eBOX;
            hit_data.index = i;
            hit_something = true;
        }
    }

    return hit_something;
}

__device__ const int MAX_DEPTH = 5;
__device__ const float3 WHITE = {1.f, 1.f, 1.f};
__device__ const float3 BLACK = {0.f, 0.f, 0.f};

template<int depth>
__device__ float3 get_color_for(DRay ray, DSphere* spheres, int sphere_count, DBox* boxes, int box_count, size_t* raycount)
{
    //
    // check for hits
    //
    bool hitspheres = false;
    bool hitboxes = false;
    DIntersection hit_data;
    hit_data.t = FLT_MAX;

    hitspheres = intersect_spheres(ray, spheres, sphere_count, hit_data);
    hitboxes = intersect_boxes(ray, boxes, box_count, hit_data);

    ++(*raycount);

    //
    // return color or continue
    //
    if (hitspheres || hitboxes)
    {
        //
        // debug - show normals
        //
        //return .5f * (1.f + normalize(hit_data.normal));

        if (hit_data.type == eSPHERE)
        {
            spheres[hit_data.index].hit_data(ray, hit_data);
        }
        else
        {
            boxes[hit_data.index].hit_data(ray, hit_data);
        }

        DRay scattered;
        float3 attenuation;
        float3 emission;
        if (hit_data.material && hit_data.material->scatter(ray, hit_data, attenuation, emission, scattered))
        {
            return emission + attenuation * get_color_for<depth + 1>(scattered, spheres, sphere_count, boxes, box_count, raycount);
        }
        else
        {
            return emission;
        }
    }

    //return WHITE;
    return BLACK;
}

template<>
__device__ float3 get_color_for<MAX_DEPTH>(DRay ray, DSphere* spheres, int sphere_count, DBox* boxes, int box_count, size_t* raycount)
{
    //return WHITE;
    return BLACK;
}

__global__ void raytrace(int width, int height, int samples, float3* pixels, size_t* raycount)
{
    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int j = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (i >= width || j >= height)
    {
        return;
    }

    hiprand_init(clock64(), i, 0, &curand_ctx);

    //
    // scene definition
    //
#define CORNELL 0
#if CORNELL
    __shared__ DSphere spheres[3];
    int spherecount = array_size(spheres);
    spheres[0].center = { 130 + 82.5 - 25, 215, 65 + 82.5 - 25 };
    spheres[0].radius = 50.f;
    spheres[0].material.type = eDIELECTRIC;
    spheres[0].material.ior = 1.5f;
    //spheres[0].material.type = eEMISSIVE;
    //spheres[0].material.albedo = { 15.f, 15.f, 15.f };

    spheres[1].center = { 265 + 82.5 + 35, 400, 295 + 82.5 - 35 };
    spheres[1].radius = 70.f;
    spheres[1].material.type = eMETAL;
    spheres[1].material.albedo = { .8f, .85f, .88f };
    spheres[1].material.roughness = .0f;
    //spheres[1].material.type = eEMISSIVE;
    //spheres[1].material.albedo = { 15.f, 15.f, 15.f };
    
    spheres[2].center = { 265 + 82.5 + 15, 30, 80 };
    spheres[2].radius = 30.f;
    spheres[2].material.type = eMETAL;
    spheres[2].material.albedo = { 1.f, .71f, .29f };
    spheres[2].material.roughness = .05f;
    //spheres[2].material.type = eEMISSIVE;
    //spheres[2].material.albedo = { 15.f, 15.f, 15.f };


    __shared__ DBox boxes[8];
    int boxcount = array_size(boxes);
    // light
    boxes[0].min_limit = { 213.f, 554.f, 227.f };
    boxes[0].max_limit = { 343.f, 555.f, 332.f };
    boxes[0].rot = { .0f, .0f, .0f };
    boxes[0].material.type = eEMISSIVE;
    boxes[0].material.albedo = { 15.f, 15.f, 15.f };
    // green side
    boxes[1].min_limit = { 555.f,    .0f, 0.f};
    boxes[1].max_limit = { 555.1f, 555.f, 555.f};
    boxes[1].rot = { .0f, .0f, .0f };
    boxes[1].material.type = eLAMBERTIAN;
    boxes[1].material.albedo = { 0.12f, 0.45f, .15f };
    // red side
    boxes[2].min_limit = { -0.1f,   .0f, 0.f };
    boxes[2].max_limit = {  .0f, 555.f, 555.f };
    boxes[2].rot = { .0f, .0f, .0f };
    boxes[2].material.type = eLAMBERTIAN;
    boxes[2].material.roughness = .0f;
    boxes[2].material.albedo = { 0.65f, .05f, .05f };
    // floor
    boxes[3].min_limit = { .0f,    -.1f, 0.f };
    boxes[3].max_limit = { 555.f, 0.f, 555.f };
    boxes[3].rot = { .0f, .0f, .0f };
    boxes[3].material.type = eLAMBERTIAN;
    boxes[3].material.albedo = { 0.73f, .73f, .73f };
    // roof
    boxes[4].min_limit = { .0f,    555.f, 0.f };
    boxes[4].max_limit = { 555.f, 555.1f, 555.f };
    boxes[4].rot = { .0f, .0f, .0f };
    //boxes[4].material.type = eEMISSIVE;
    //boxes[4].material.albedo = { 1.f, 1.f, 1.f };
    boxes[4].material.type = eLAMBERTIAN;
    boxes[4].material.albedo = { 0.73f, .73f, .73f };
    //back
    boxes[5].min_limit = { .0f,    .0f, 554.9f };
    boxes[5].max_limit = { 555.f, 555.f, 555.f };
    boxes[5].rot = { .0f, .0f, .0f };
    boxes[5].material.type = eLAMBERTIAN;
    boxes[5].material.albedo = { 0.73f, .73f, .73f };
    // higher block
    boxes[6].min_limit = { 265.f,   .0f, 295.f };
    boxes[6].max_limit = { 430.f, 330.f, 460.f };
    boxes[6].rot = { .0f, 15.0f, .0f };
    boxes[6].material.type = eLAMBERTIAN;
    boxes[6].material.albedo = { 0.73f, .73f, .73f };
    // lower block
    boxes[7].min_limit = { 130.f,   .0f, 65.f };
    boxes[7].max_limit = { 295.f, 165.f, 230.f };
    boxes[7].rot = { .0f, -18.f, .0f };
    boxes[7].material.type = eLAMBERTIAN;
    boxes[7].material.albedo = { 0.73f, .73f, .73f };

    //
    // camera setup
    //
    const float fov = radians(40.f);
    const float aspect = width / fmax(1.f, static_cast<float>(height));
    const float3 center{ 278.f, 278.f, -800.f };
    const float3 lookat{ 278.f, 278.f, 0.f };
    const float3 vup{ 0.f, 1.f, 0.f };

#else
    __shared__ DSphere spheres[8];
    int spherecount = array_size(spheres);
    spheres[0].center = { 0.f, 0.f, -1.f };
    spheres[0].radius = .5f;
    spheres[0].material.type = eLAMBERTIAN;
    spheres[0].material.albedo = { 0.1f, 0.2f, 0.5f };

    spheres[1].center = { 0.f, -100.5f, -1.f };
    spheres[1].radius = 100.f;
    spheres[1].material.type = eLAMBERTIAN;
    spheres[1].material.albedo = { 0.2f, 0.2f, 0.2f };

    spheres[2].center = { 1.f, 0.f, -1.f };
    spheres[2].radius = .5f;
    spheres[2].material.type = eMETAL;
    spheres[2].material.albedo = { .8f, .85f, .88f };
    spheres[2].material.roughness = .0f;

    spheres[3].center = { -1.f, 0.f, -1.f };
    spheres[3].radius = .5f;
    spheres[3].material.type = eDIELECTRIC;
    spheres[3].material.ior = 1.5f;
    //spheres[3].material.type = eMETAL;
    //spheres[3].material.albedo = { .8f, .85f, .88f };
    //spheres[3].material.roughness = .05f;

    spheres[4].center = { 0.f, 150.f, -1.f };
    spheres[4].radius = 100.f;
    spheres[4].material.type = eEMISSIVE;
    spheres[4].material.albedo = { 2.f, 2.f, 2.f };

    spheres[5].center = { 0.f, 0.f, 0.f };
    spheres[5].radius = .2f;
    //spheres[5].material.type = eEMISSIVE;
    //spheres[5].material.albedo = { 2.f, 2.f, 2.f };
    spheres[5].material.type = eDIELECTRIC;
    spheres[5].material.ior = 1.5f;

    spheres[6].center = { 0.f, 1.f, -1.5f };
    spheres[6].radius = .3f;
    spheres[6].material.type = eMETAL;
    spheres[6].material.albedo = { 1.f, .71f, .29f };
    spheres[6].material.roughness = .05f;

    spheres[7].center = { 0.f, 0.f, -2.5f };
    spheres[7].radius = .5f;
    spheres[7].material.type = eLAMBERTIAN;
    spheres[7].material.albedo = { .85f, .05f, .02f };

    __shared__ DBox boxes[1];
    int boxcount = array_size(boxes);
    boxes[0].min_limit = { -2.f, 0.f, -3.1f };
    boxes[0].max_limit = { 2.f, 2.f, -3.f };
    boxes[0].material.type = eLAMBERTIAN;
    boxes[0].material.albedo = { .05f, .85f, .02f };

    //
    // camera setup
    //
    const float fov = radians(60.f);
    const float aspect = width / fmax(1.f, static_cast<float>(height));
    const float3 center{ -.5f, 1.2f, 1.5f };
    const float3 lookat{ 0.f, 0.f, -1.f };
    const float3 vup{ 0.f, 1.f, 0.f };
#endif

    float height_2 = tanf(fov / 2.f);
    float width_2 = height_2 * aspect;

    float3 w = normalize(center - lookat);
    float3 u = normalize(cross(vup, w));
    float3 v = cross(w, u);

    float3 horizontal = 2.f * width_2 * u;
    float3 vertical = 2.f * height_2 * v;
    float3 origin = center - width_2 * u - height_2 * v - w;
    
    //
    // main loop
    //
    size_t raycount_inst = 0;
    float3 color{ .0f, .0f, .0f };
    for (int sample = 0; sample < samples; ++sample)
    {
        float s = ((i + cuda_fastrand()) / static_cast<float>(width));
        float t = ((j + cuda_fastrand()) / static_cast<float>(height));

        DRay ray;
        ray.origin = center;
        ray.direction = normalize(origin + s * horizontal + t * vertical - center);

        color += get_color_for<0>(ray, spheres, spherecount, boxes, boxcount, &raycount_inst);
    }

    atomicAdd(raycount, raycount_inst);
    
    //
    // debug output if needed
    //
    //color.x = s;
    //color.y = t;
    //color.z = .0f;
    
    float3& pixel = *(&pixels[j * width + i]);
    //pixel.x = color.x;
    //pixel.y = color.y;
    //pixel.z = color.z;

    atomicAdd(&pixel.x, color.x);
    atomicAdd(&pixel.y, color.y);
    atomicAdd(&pixel.z, color.z);

    // just to be sure we're running
    if (i == 0 && j == 0 && blockIdx.z == 0) { CUDALOG("[CUDA] running kernel...\n"); }
}

//
// IFace for raytracer.cpp
// 
extern "C" void cuda_trace(int w, int h, int ns, float* out_buffer, size_t& raycount)
{
    // ensure output buffer is properly zeroed
    memset(out_buffer, 0, w * h * sizeof(float3));

    const int MAX_GPU = 32;
    int num_gpus = 0;
    checkCudaErrors(hipGetDeviceCount(&num_gpus));

    hipDeviceProp_t gpu_properties[MAX_GPU];
    for (int i = 0; i < num_gpus; i++)
    {
        checkCudaErrors(hipGetDeviceProperties(&gpu_properties[i], i));

        CUDALOG("Device %d (%s):\n\t%d threads\n\tblocksize: %dx%dx%d\n\tshmem per block: %lu Kb\n\tgridsize: %dx%dx%d\n\n",
               i,
               gpu_properties[i].name,
               gpu_properties[i].maxThreadsPerBlock,
               gpu_properties[i].maxThreadsDim[0], gpu_properties[i].maxThreadsDim[1], gpu_properties[i].maxThreadsDim[2],
               static_cast<unsigned long>(gpu_properties[i].sharedMemPerBlock / 1024.f),
               gpu_properties[i].maxGridSize[0], gpu_properties[i].maxGridSize[1], gpu_properties[i].maxGridSize[2]);
    }

    CUDALOG("image is %dx%d (%d samples desired)\n", w, h, ns);

#if CUDA_USE_STREAMS
    hipStream_t d_stream[MAX_GPU];
#endif

    num_gpus = min(num_gpus, MAX_GPU);

    size_t* d_raycount[MAX_GPU];
    float3* d_output_cuda[MAX_GPU];
    float* h_output_cuda[MAX_GPU];

#if CUDA_USE_MULTIGPU
    for (int i = num_gpus - 1; i >= 0; --i)
    {
#else
    {
        int i = num_gpus - 1;
#endif
        checkCudaErrors(hipSetDevice(i));

#if CUDA_USE_STREAMS
        checkCudaErrors(hipStreamCreateWithFlags(&d_stream[i], hipStreamNonBlocking));
#endif

        checkCudaErrors(hipMalloc((void**)&d_output_cuda[i], w * h * sizeof(float3)));
        checkCudaErrors(hipMemset((void*)d_output_cuda[i], 0, w * h * sizeof(float3)));

        checkCudaErrors(hipMalloc((void**)&d_raycount[i], sizeof(size_t)));

        checkCudaErrors(hipHostMalloc((void**)&h_output_cuda[i], w * h * sizeof(float3)));
    }

#if CUDA_USE_MULTIGPU
    for (int i = num_gpus - 1; i >= 0; --i)
    {
#else
    {
        int i = num_gpus - 1;
#endif
        checkCudaErrors(hipSetDevice(i));

        int threads_per_row = sqrt(gpu_properties[i].maxThreadsPerBlock);
        int block_depth = 1;

#if CUDA_USE_MULTIGPU
        int gpu_split = num_gpus;
#else
        int gpu_split = 1;
#endif

        dim3 dimBlock(threads_per_row, threads_per_row, 1);
        dim3 dimGrid(w / dimBlock.x + 1, h / dimBlock.y + 1, block_depth);
        
        CUDALOG("raytrace<<<(%d,%d,%d), (%d,%d,%d)>>> on gpu %d\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z, i);
#if CUDA_USE_STREAMS
        raytrace<<<dimGrid, dimBlock, 0, d_stream[i]>>>(w, h, ns / block_depth / gpu_split, d_output_cuda[i], d_raycount[i]);
#else
        raytrace<<<dimGrid, dimBlock>>>(w, h, ns / block_depth / gpu_split, d_output_cuda[i], d_raycount[i]);
#endif
    }

#if CUDA_USE_MULTIGPU
    for (int i = num_gpus - 1; i >= 0; --i)
    {
#else
    {
        int i = num_gpus - 1;
#endif
        checkCudaErrors(hipSetDevice(i));

#if CUDA_USE_STREAMS
        checkCudaErrors(hipMemcpyAsync(h_output_cuda[i], d_output_cuda[i], w * h * sizeof(float3), hipMemcpyDeviceToHost, d_stream[i]));

        checkCudaErrors(hipStreamSynchronize(d_stream[i]));
#else
        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipMemcpy(h_output_cuda[i], d_output_cuda[i], w * h * sizeof(float3), hipMemcpyDeviceToHost));
#endif
      
#if CUDA_USE_MULTIGPU
        for (int j = 0; j < w * h * 3; ++j)
        {
            out_buffer[j] += h_output_cuda[i][j];
        }
#else
        memcpy(out_buffer, h_output_cuda[i], w * h * 3 * sizeof(float));
#endif

        size_t tmp;
        checkCudaErrors(hipMemcpy(&tmp, d_raycount[i], sizeof(size_t), hipMemcpyDeviceToHost));
        raycount += tmp;

        CUDALOG("cuda compute (%d/%d) completed!\n", i, num_gpus - 1);

        checkCudaErrors(hipFree(d_raycount[i]));
        checkCudaErrors(hipFree(d_output_cuda[i]));
        checkCudaErrors(hipHostFree(h_output_cuda[i]));

#if CUDA_USE_STREAMS
        checkCudaErrors(hipStreamDestroy(d_stream[i]));
#endif
    }
}
